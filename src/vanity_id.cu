#include <hip/hip_runtime.h>

#include <stdint.h>

// Constants for the character mapping
__constant__ char MAPPING[16] = {'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p'};

// SHA-256 constants
__constant__ uint32_t K[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

// SHA-256 helper functions
__device__ uint32_t rotr(uint32_t x, uint32_t n)
{
  return (x >> n) | (x << (32 - n));
}

__device__ uint32_t ch(uint32_t x, uint32_t y, uint32_t z)
{
  return (x & y) ^ (~x & z);
}

__device__ uint32_t maj(uint32_t x, uint32_t y, uint32_t z)
{
  return (x & y) ^ (x & z) ^ (y & z);
}

__device__ uint32_t sigma0(uint32_t x)
{
  return rotr(x, 2) ^ rotr(x, 13) ^ rotr(x, 22);
}

__device__ uint32_t sigma1(uint32_t x)
{
  return rotr(x, 6) ^ rotr(x, 11) ^ rotr(x, 25);
}

__device__ uint32_t gamma0(uint32_t x)
{
  return rotr(x, 7) ^ rotr(x, 18) ^ (x >> 3);
}

__device__ uint32_t gamma1(uint32_t x)
{
  return rotr(x, 17) ^ rotr(x, 19) ^ (x >> 10);
}

// Generate key data from counter
__device__ void generate_key_data(uint64_t counter, unsigned char *key_data)
{
  // Clear the array
  for (int i = 0; i < 32; i++)
  {
    key_data[i] = 0;
  }

  // Use counter as base (little endian)
  for (int i = 0; i < 8; i++)
  {
    key_data[i] = (counter >> (i * 8)) & 0xFF;
  }

  // Fill remaining bytes with derived values
  for (int i = 8; i < 32; i++)
  {
    key_data[i] = ((counter >> (i % 8)) ^ i) & 0xFF;
  }
}

// SHA-256 implementation
__device__ void sha256(const unsigned char *data, unsigned char *hash)
{
  uint32_t h[8] = {
      0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
      0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19};

  // Prepare message schedule
  uint32_t w[64];

  // Copy input data to first 16 words (big endian)
  for (int i = 0; i < 8; i++)
  {
    w[i] = (data[i * 4] << 24) | (data[i * 4 + 1] << 16) | (data[i * 4 + 2] << 8) | data[i * 4 + 3];
  }

  // Padding: append 1 bit followed by zeros, then length
  w[8] = 0x80000000; // First padding bit
  for (int i = 9; i < 14; i++)
  {
    w[i] = 0;
  }
  w[14] = 0;   // High 32 bits of length (always 0 for our 32-byte input)
  w[15] = 256; // Low 32 bits of length (32 bytes = 256 bits)

  // Extend the first 16 words into the remaining 48 words
  for (int i = 16; i < 64; i++)
  {
    w[i] = gamma1(w[i - 2]) + w[i - 7] + gamma0(w[i - 15]) + w[i - 16];
  }

  // Main loop
  uint32_t a = h[0], b = h[1], c = h[2], d = h[3];
  uint32_t e = h[4], f = h[5], g = h[6], h_val = h[7];

  for (int i = 0; i < 64; i++)
  {
    uint32_t t1 = h_val + sigma1(e) + ch(e, f, g) + K[i] + w[i];
    uint32_t t2 = sigma0(a) + maj(a, b, c);
    h_val = g;
    g = f;
    f = e;
    e = d + t1;
    d = c;
    c = b;
    b = a;
    a = t1 + t2;
  }

  // Add this chunk's hash to result
  h[0] += a;
  h[1] += b;
  h[2] += c;
  h[3] += d;
  h[4] += e;
  h[5] += f;
  h[6] += g;
  h[7] += h_val;

  // Convert to bytes (big endian)
  for (int i = 0; i < 8; i++)
  {
    hash[i * 4] = (h[i] >> 24) & 0xFF;
    hash[i * 4 + 1] = (h[i] >> 16) & 0xFF;
    hash[i * 4 + 2] = (h[i] >> 8) & 0xFF;
    hash[i * 4 + 3] = h[i] & 0xFF;
  }
}

// Check if hash matches prefix
__device__ bool hash_matches_prefix(const unsigned char *hash, const unsigned char *prefix, uint32_t prefix_len)
{
  if (prefix_len == 0)
    return true;

  uint32_t full_bytes = prefix_len / 2;

  // Check full bytes
  for (uint32_t byte_idx = 0; byte_idx < full_bytes; byte_idx++)
  {
    unsigned char hash_byte = hash[byte_idx];
    unsigned char expected_high = prefix[byte_idx * 2];
    unsigned char expected_low = prefix[byte_idx * 2 + 1];

    unsigned char actual_high = MAPPING[(hash_byte >> 4) & 0x0F];
    unsigned char actual_low = MAPPING[hash_byte & 0x0F];

    if (actual_high != expected_high || actual_low != expected_low)
    {
      return false;
    }
  }

  // Handle odd-length prefix
  if (prefix_len % 2 == 1)
  {
    unsigned char hash_byte = hash[full_bytes];
    unsigned char expected_char = prefix[prefix_len - 1];
    unsigned char actual_char = MAPPING[(hash_byte >> 4) & 0x0F];

    if (actual_char != expected_char)
    {
      return false;
    }
  }

  return true;
}

// Main CUDA kernel
__global__ void vanity_search(
    uint32_t *results,           // Output: [found_flag, counter_low, counter_high, key_data...]
    const unsigned char *prefix, // Input: prefix to search for
    uint32_t prefix_len,         // Input: length of prefix
    uint64_t start_counter       // Input: starting counter value
)
{
  uint64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t counter = start_counter + thread_id;

  // Generate key data
  unsigned char key_data[32];
  generate_key_data(counter, key_data);

  // Compute SHA-256 hash
  unsigned char hash[32];
  sha256(key_data, hash);

  // Check if it matches the prefix
  if (hash_matches_prefix(hash, prefix, prefix_len))
  {
    // Atomically set found flag
    uint32_t old = atomicCAS(&results[0], 0, 1);
    if (old == 0)
    {
      // We're the first to find a match, store the result
      // Split 64-bit counter into two 32-bit values
      results[1] = (uint32_t)(counter & 0xFFFFFFFF);         // Low 32 bits
      results[2] = (uint32_t)((counter >> 32) & 0xFFFFFFFF); // High 32 bits

      // Store key data (convert to uint32 for easier transfer)
      for (int i = 0; i < 8; i++)
      {
        uint32_t key_chunk = 0;
        for (int j = 0; j < 4; j++)
        {
          key_chunk |= ((uint32_t)key_data[i * 4 + j]) << (j * 8);
        }
        results[3 + i] = key_chunk;
      }
    }
  }
}

// C interface for Rust FFI
extern "C"
{
  // Initialize CUDA and return device properties
  int cuda_init(int *max_threads_per_block, char *device_name, int name_len);

  // Search for vanity ID using CUDA
  int cuda_search_vanity_id(
      const char *prefix,
      int prefix_len,
      uint64_t start_counter,
      uint64_t batch_size,
      uint32_t *results // [found_flag, counter_low, counter_high, key_data_as_8_u32s]
  );

  // Cleanup CUDA resources
  void cuda_cleanup();
}

// Global device pointers
static unsigned char *d_prefix = nullptr;
static uint32_t *d_results = nullptr;
static bool cuda_initialized = false;

int cuda_init(int *max_threads_per_block, char *device_name, int name_len)
{
  if (cuda_initialized)
  {
    return 0; // Already initialized
  }

  // Check for CUDA devices
  int device_count;
  hipError_t error = hipGetDeviceCount(&device_count);
  if (error != hipSuccess || device_count == 0)
  {
    return -1; // No CUDA devices found
  }

  // Get device properties
  hipDeviceProp_t prop;
  error = hipGetDeviceProperties(&prop, 0);
  if (error != hipSuccess)
  {
    return -2; // Failed to get device properties
  }

  // Set device
  error = hipSetDevice(0);
  if (error != hipSuccess)
  {
    return -3; // Failed to set device
  }

  // Copy device name
  int copy_len = (name_len - 1 < strlen(prop.name)) ? name_len - 1 : strlen(prop.name);
  strncpy(device_name, prop.name, copy_len);
  device_name[copy_len] = '\0';

  *max_threads_per_block = prop.maxThreadsPerBlock;

  // Allocate device memory for results (11 uint32_t values)
  error = hipMalloc(&d_results, 11 * sizeof(uint32_t));
  if (error != hipSuccess)
  {
    return -4; // Failed to allocate results buffer
  }

  // Allocate device memory for prefix (max 64 characters should be enough)
  error = hipMalloc(&d_prefix, 64);
  if (error != hipSuccess)
  {
    hipFree(d_results);
    return -5; // Failed to allocate prefix buffer
  }

  cuda_initialized = true;
  return 0; // Success
}

int cuda_search_vanity_id(
    const char *prefix,
    int prefix_len,
    uint64_t start_counter,
    uint64_t batch_size,
    uint32_t *results)
{
  if (!cuda_initialized)
  {
    return -1; // CUDA not initialized
  }

  // Copy prefix to device
  hipError_t error = hipMemcpy(d_prefix, prefix, prefix_len, hipMemcpyHostToDevice);
  if (error != hipSuccess)
  {
    return -2; // Failed to copy prefix
  }

  // Initialize results buffer to zero
  error = hipMemset(d_results, 0, 11 * sizeof(uint32_t));
  if (error != hipSuccess)
  {
    return -3; // Failed to initialize results
  }

  // Calculate grid and block dimensions
  int threads_per_block = 256; // Good default for most GPUs
  int blocks = (batch_size + threads_per_block - 1) / threads_per_block;

  // Limit the number of blocks to avoid excessive GPU usage
  const int max_blocks = 65535; // CUDA grid limit
  if (blocks > max_blocks)
  {
    blocks = max_blocks;
  }

  // Launch kernel
  vanity_search<<<blocks, threads_per_block>>>(
      d_results,
      d_prefix,
      prefix_len,
      start_counter);

  // Wait for kernel to complete
  error = hipDeviceSynchronize();
  if (error != hipSuccess)
  {
    return -4; // Kernel execution failed
  }

  // Copy results back to host
  error = hipMemcpy(results, d_results, 11 * sizeof(uint32_t), hipMemcpyDeviceToHost);
  if (error != hipSuccess)
  {
    return -5; // Failed to copy results
  }

  return 0; // Success
}

void cuda_cleanup()
{
  if (cuda_initialized)
  {
    if (d_prefix)
    {
      hipFree(d_prefix);
      d_prefix = nullptr;
    }
    if (d_results)
    {
      hipFree(d_results);
      d_results = nullptr;
    }
    hipDeviceReset();
    cuda_initialized = false;
  }
}
